#include "hip/hip_runtime.h"
/**
 * 
 * Copyright (C) 2021-2022, David Castells-Rufas <david.castells@uab.cat>, 
 * Universitat Autonoma de Barcelona  
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

// @todo By now, we use statically compiled CUDA code, we should try the NVRTC API to run-time compile the kernel as we do with OpenCL

#define TILE_LEN        3
//#define WORKGROUP_SIZE  120
#define WORKGROUP_SIZE  32

#define SHARED_STORE


/* 
 * File:   CUDAWavefrontOriginal2Cols.cpp
 * Author: dcr
 * 
 * Created on April 24, 2022, 11:30 AM
 */

#include "CUDAWavefrontOriginal2Cols.h"
#include "utils.h"

#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <string.h>
#include <new>


#define CARTESIAN_TO_INDEX(y, x, w)		((y)*(w) + (x))
//#define POLAR_D_TO_INDEX(d, r, w)			CARTESIAN_TO_INDEX(POLAR_D_TO_CARTESIAN_Y((d), (r)),POLAR_D_TO_CARTESIAN_X((d), (r)),w)
#define POLAR_D_TO_CARTESIAN_Y(d,r)		((((d) >= 0)? 0: -(d)) + (r))
#define POLAR_D_TO_CARTESIAN_X(d,r)		((((d) >= 0)? (d): 0) + (r))

#define CARTESIAN_TO_POLAR_D_D(y, x)		((x)-(y))
#define CARTESIAN_TO_POLAR_D_R(y, x)		(((y)>(x))? (x) : (y))


//#define POLAR_W_TO_INDEX(d, r)		((d)+m_k + (((r)%(2*tileLen)) * (2*m_k+1)))
#define POLAR_W_TO_INDEX(d, r)		((d)+m_k + (((r)%2) * (2*m_k+1)))
//#define INDEX_TO_POLAR_W_D(idx, r)      ((idx) - (m_k) - ((r)%2)*(2*m_k+1))
//#define INDEX_TO_POLAR_W_D(idx, r)      ((idx) - (m_k) - ((r)%(2*tileLen))*(2*m_k+1))


#define POLAR_W_TO_CARTESIAN_Y(d,r)		((((d) >= 0)? -(d) : 0 ) + (r))
#define POLAR_W_TO_CARTESIAN_X(d,r)		((((d) >= 0)? 0 : (d)) + (r))

#define CARTESIAN_TO_POLAR_W_D(y, x)		((x)-(y))
#define CARTESIAN_TO_POLAR_W_R(y, x)		(((y)>(x))? (y) : (x))


#define POLAR_LOCAL_W_TO_INDEX(d, r, tl) ((r) >= (tl))? (2*((r)-(tl))*(tl)) - ((r)-(tl))*((r)-(tl)) + (2*(tl)-(r)-1) + (d) + (tl)*(tl) : ((r)*(r)+(r)+d)

// WARNING !!!! This is harcoded here, where should it be ?
// WARNING !!!! This is harcoded here, where should it be ?
// WARNING !!!! This is harcoded here, where should it be ?
// WARNING !!!! This is harcoded here, where should it be ?
#define SHARED_STORE

#ifdef GLOBAL_STORE
    #define LOCAL_TILE_TYPE     __private
#endif

#ifdef REGISTER_STORE
    // we define a tile with len 3
    typedef struct
    {
        long r0_d0;
        long r1_dm1;
        long r1_d0;
        long r1_d1;
        long r2_dm2;
        long r2_dm1;
        long r2_d0;
        long r2_d1;
        long r2_d2;
        long r3_dm2;
        long r3_dm1;
        long r3_d0;
        long r3_d1;
        long r3_d2;
        long r4_dm1;
        long r4_d0;
        long r4_d1;
        long r5_d0;
    } LOCAL_TILE;
    #define LOCAL_TILE_PTR      LOCAL_TILE*
#endif

#ifdef SHARED_STORE
    #define LOCAL_TILE_TYPE     __shared__
    #define LOCAL_TILE_PTR      INT_TYPE*
#endif


extern int verbose;
extern int gPid;
extern int gDid;
extern int gTileLen;
extern int gMeasureIterationTime;
extern int gEnqueuedInvocations;
extern int gWorkgroupSize;

__forceinline__ __device__
INT_TYPE extend(const char* P, const char* T, INT_TYPE m, INT_TYPE n, INT_TYPE pi, INT_TYPE ti)
{
    long e = 0;

    while (pi < m && ti < n)
    {
            if (P[pi] != T[ti])
                    return e;
            e++;
            pi++;
            ti++;
    }

    return e;
}

__forceinline__ __device__
int polarExistsInW(INT_TYPE d, INT_TYPE r)
{
    int ret =  abs(d) <= r;
    //printf("polar exist in W (%ld, %ld) = %d\n", d, r, ret);
    return ret;
}

__forceinline__ __device__ 
int isInLocalBlock(int ld, int lr)
{
    if (lr < 0) return 0;
    if (lr >= 2*TILE_LEN) return 0;
    
    INT_TYPE dec_lr = 2*TILE_LEN -1 - lr;
    INT_TYPE min_lr = min(lr, dec_lr);
    
    if (abs(ld) > min_lr) return 0;
    
    return 1;
}


__forceinline__ __device__
int isInLocalBlockBoundary(int ld, int lr, int tileLen)
{
    int col = lr + abs(ld);
    const int col1 = 2*TILE_LEN - 1;
    const int col2 = col1-1;
    
    return ((col == col1) || (col == col2));
}

#define WRITE_W(d,r, v)         writeToW(m_W, localW, (d), (r), (v), m_k, tileLen, ld, lr)
#define READ_W(d,r, ld, lr)     (readFromW(m_W, localW, (d), (r), m_k, tileLen, ld, lr))

#ifdef REGISTER_STORE
    void inline __attribute__((always_inline)) 
    writeLocalW(LOCAL_TILE_PTR localW, INT_TYPE d, INT_TYPE r, INT_TYPE v)
    {
        //printf("local_WR(%ld,%ld) = %ld\n", d, r, v);
        // max 8
        switch ((r<<3)|(d+2))
        {
            case (0<<3)|(0+2):  localW->r0_d0 = v; break;
            case (1<<3)|(-1+2): localW->r1_dm1 = v; break;
            case (1<<3)|(0+2):  localW->r1_d0 = v; break;
            case (1<<3)|(1+2):  localW->r1_d1 = v; break;
            case (2<<3)|(-2+2): localW->r2_dm2 = v; break;
            case (2<<3)|(-1+2): localW->r2_dm1 = v; break;
            case (2<<3)|(0+2):  localW->r2_d0 = v; break;
            case (2<<3)|(1+2):  localW->r2_d1 = v; break;
            case (2<<3)|(2+2):  localW->r2_d2 = v; break;
            case (3<<3)|(-2+2): localW->r3_dm2 = v; break;
            case (3<<3)|(-1+2): localW->r3_dm1 = v; break;
            case (3<<3)|(0+2):  localW->r3_d0 = v; break;
            case (3<<3)|(1+2):  localW->r3_d1 = v; break;
            case (3<<3)|(2+2):  localW->r3_d2 = v; break;
            case (4<<3)|(-1+2): localW->r4_dm1 = v; break;
            case (4<<3)|(0+2):  localW->r4_d0 = v; break;
            case (4<<3)|(1+2):  localW->r4_d1 = v; break;
            case (5<<3)|(0+2):  localW->r5_d0 = v; break;
            default:
                printf("WRITE ERROR r:%d d:%d\n", r, d);

        }
        
        //printf("localW.r0_d0 = %ld\n", localW->r0_d0 );

    }

    void inline __attribute__((always_inline)) 
    writeToW(INT_TYPE* m_W, LOCAL_TILE_PTR localW, INT_TYPE d, INT_TYPE r, INT_TYPE v, INT_TYPE m_k, int tileLen, int ld, int lr)
    {    
        writeLocalW(localW, ld, lr, v);
        //int lidx = POLAR_LOCAL_W_TO_INDEX(ld, lr, tileLen);
        //localW[lidx] = v;

        int inBoundary = isInLocalBlockBoundary(ld, lr);
        
    #ifdef DEBUG
        printf("WR(%ld, %ld, %d) -> local WR(%d, %d, %d) -> WR  = %ld (in boundary: %d)\n", 
                d, r, tileLen, 
                ld, lr, tileLen, v, inBoundary);
    #endif
        
        if (inBoundary)
        {
            m_W[POLAR_W_TO_INDEX(d, r)] = v;
        }
    }
#else
    __forceinline__ __device__
    INT_TYPE readFromW(INT_TYPE* m_W, LOCAL_TILE_PTR localW, INT_TYPE d, INT_TYPE r, INT_TYPE m_k, int tileLen, int ld, int lr)
    {
        int isInLocal = isInLocalBlock(ld, lr);
        
        if (isInLocal)
        {
            int lidx = POLAR_LOCAL_W_TO_INDEX(ld, lr, tileLen);
            INT_TYPE lv = localW[lidx];

    #ifdef DEBUG
            printf("RD(%ld, %ld, %d) -> local RD(%d, %d, %d) -> RD idx(%d) = %ld\n", d, r, tileLen, ld, lr, tileLen, lidx, lv);
    #endif
        
            return lv;
        }
        else
        {
            INT_TYPE gv = m_W[POLAR_W_TO_INDEX(d, r)];
            
    #ifdef DEBUG
            printf("RD(%ld, %ld, %d) ->  = %ld\n", d, r, tileLen, gv);
    #endif    
            return gv;
        }
    }
#endif


#ifdef REGISTER_STORE
    INT_TYPE inline __attribute__((always_inline)) 
    readLocalW(LOCAL_TILE_PTR localW, INT_TYPE d, INT_TYPE r)
    {
        INT_TYPE v; 
        
        // max 8
        switch ((r<<3)|(d+2))
        {
            case (0<<3)|(0+2):  v = localW->r0_d0; break;
            case (1<<3)|(-1+2): v = localW->r1_dm1; break;
            case (1<<3)|(0+2):  v = localW->r1_d0; break;
            case (1<<3)|(1+2):  v = localW->r1_d1; break;
            case (2<<3)|(-2+2): v = localW->r2_dm2; break;
            case (2<<3)|(-1+2): v = localW->r2_dm1; break;
            case (2<<3)|(0+2):  v = localW->r2_d0; break;
            case (2<<3)|(1+2):  v = localW->r2_d1; break;
            case (2<<3)|(2+2):  v = localW->r2_d2; break;
            case (3<<3)|(-2+2): v = localW->r3_dm2; break;
            case (3<<3)|(-1+2): v = localW->r3_dm1; break;
            case (3<<3)|(0+2):  v = localW->r3_d0; break;
            case (3<<3)|(1+2):  v = localW->r3_d1; break;
            case (3<<3)|(2+2):  v = localW->r3_d2; break;
            case (4<<3)|(-1+2): v = localW->r4_dm1; break;
            case (4<<3)|(0+2):  v = localW->r4_d0; break;
            case (4<<3)|(1+2):  v = localW->r4_d1; break;
            case (5<<3)|(0+2):  v = localW->r5_d0; break;
            default:
                printf("READ ERROR r:%d d:%d\n", r, d);
        }
        

        //printf("local_RD(%ld,%ld) = %ld\n", d, r, v);
        return v;
    }
    
    INT_TYPE inline __attribute__((always_inline)) 
    readFromW(INT_TYPE* m_W, LOCAL_TILE_PTR localW, INT_TYPE d, INT_TYPE r, INT_TYPE m_k, int tileLen, int ld, int lr)
    {
        int isInLocal = isInLocalBlock(ld, lr);
        
        if (isInLocal)
        {
            INT_TYPE lv = readLocalW(localW, ld, lr);
            

    #ifdef DEBUG
        
            printf("RD(%ld, %ld, %d) -> local RD(%d, %d, %d) -> RD idx(%d) = %ld\n", 
                d, r, tileLen, 
                ld, lr, tileLen, lidx, lv);
    #endif
        
            return lv;
        }
        else
        {
            INT_TYPE gv = m_W[POLAR_W_TO_INDEX(d, r)];
            
    #ifdef DEBUG
            printf("RD(%ld, %ld, %d) ->  = %ld\n", 
                d, r, tileLen, gv);
    #endif    
            return gv;
        }
    }
#else
    __forceinline__ __device__
    void writeToW(INT_TYPE* m_W, INT_TYPE* localW, INT_TYPE d, INT_TYPE r, INT_TYPE v, INT_TYPE m_k, int tileLen, int ld, int lr)
    {    
        int lidx = POLAR_LOCAL_W_TO_INDEX(ld, lr, tileLen);

        localW[lidx] = v;

        int inBoundary = isInLocalBlockBoundary(ld, lr, tileLen);
        
    #ifdef DEBUG
        printf("WR(%ld, %ld, %d) -> local WR(%d, %d, %d) -> WR idx(%d) = %ld (in boundary: %d)\n", 
                d, r, tileLen, 
                ld, lr, tileLen, lidx, v, inBoundary);
    #endif
        
        if (inBoundary)
        {
            m_W[POLAR_W_TO_INDEX(d, r)] = v;
        }
    }
#endif


__forceinline__ __device__
void processCell(char* P, 
        char* T, 
        INT_TYPE m_m, 
        INT_TYPE m_n,
        INT_TYPE m_k, 
        INT_TYPE* m_W,
        INT_TYPE* p_final_d_r,
        INT_TYPE d,
        INT_TYPE r,
        int tileLen,
        LOCAL_TILE_PTR localW,
        int ld,
        int lr,
        int* doRun)
{
    INT_TYPE m_top = max2(m_m,m_n);

    // we already reached the final point in previous invocations
    if (p_final_d_r[0] >= m_top)
    {
        *doRun = 0;
        return;
    }
       
    INT_TYPE final_d = CARTESIAN_TO_POLAR_D_D(m_m, m_n);

    // early exit for useless work items
    if (!polarExistsInW(d,r))
        return;
            
    if (r == 0)
    {
        if (d == 0)
        {
            // initial case
            INT_TYPE extended = extend(P, T, m_m, m_n, 0, 0);
            WRITE_W(d, r, extended);
            
            if ((d == final_d) && extended >= m_top)
            {
                //printf("COMPLETE\n");
                p_final_d_r[0] = extended;  // furthest reaching point
                p_final_d_r[1] = r;         // at edit distance = r
                *doRun = 0;
                return;
            }
        }
        else
            WRITE_W(d, r, 0);
    }
    else
    {
        INT_TYPE diag_up = (polarExistsInW(d+1, r-1))? READ_W(d+1, r-1, ld+1, lr-1)  : 0;   // m_W[POLAR_W_TO_INDEX(d+1, r-1)] 
        INT_TYPE left = (polarExistsInW(d,r-1))? READ_W(d, r-1, ld, lr-1) : 0;              // m_W[POLAR_W_TO_INDEX(d, r-1)]
        INT_TYPE diag_down = (polarExistsInW(d-1,r-1))? READ_W(d-1, r-1, ld-1, lr-1) : 0;   // m_W[POLAR_W_TO_INDEX(d-1, r-1)]  

        //printf("u|l|r = %ld|%ld|%ld\n",  diag_up, left, diag_down);

        INT_TYPE compute;

        if (d == 0)
            compute = max3(diag_up, left+1, diag_down);
        else if (d > 0)
            compute = max3(diag_up, left+1, diag_down+1);
        else
            compute = max3(diag_up+1, left+1, diag_down);

        if ((d == final_d) && compute >= m_top)
        {
            // m_W[POLAR_W_TO_INDEX(d, r)] = compute;
            WRITE_W(d,r, compute);
            p_final_d_r[0] = compute;   // furthest reaching point
            p_final_d_r[1] = r;         // at edit distance = r
            
            // printf("W[d:%ld,r:%ld]=%ld\n", d, r, compute);
            *doRun = 0;
            return;
        }

        INT_TYPE ex = POLAR_W_TO_CARTESIAN_X(d, compute);
        INT_TYPE ey = POLAR_W_TO_CARTESIAN_Y(d, compute);

        if ((ex < m_n) && (ey < m_m))
        {
            INT_TYPE extendv = extend(P, T, m_m, m_n, ey, ex);
            INT_TYPE extended = compute + extendv;

            // m_W[POLAR_W_TO_INDEX(d, r)] = extended;
            WRITE_W(d, r, extended);

            if ((d == final_d) && extended >= m_top)
            {
                p_final_d_r[0] = extended;  // furthest reaching point
                p_final_d_r[1] = r;         // at edit distance = r

                //printf("W[d:%ld,r:%ld]=%ld\n", d, r, extended);
                *doRun = 0;
                return;
            }
        }
        else
        {
            //m_W[POLAR_W_TO_INDEX(d, r)] = compute;
            WRITE_W(d, r, compute);
            // it is impossible to assign the final result here, because it would
            // have been in the previous compute check
        }
    }

}

__global__
void wfo2cols(
        char* P, 
        char* T, 
        INT_TYPE m_m, 
        INT_TYPE m_n, 
        INT_TYPE r0, 
        INT_TYPE m_k,  
        INT_TYPE* m_W,
        INT_TYPE* p_final_d_r,
        int tileLen)
{
#ifdef GLOBAL_STORE
    LOCAL_TILE_TYPE INT_TYPE localW[2*TILE_LEN*TILE_LEN];    
#endif

#ifdef REGISTER_STORE
    LOCAL_TILE localW;
#endif
    
#ifdef SHARED_STORE
    LOCAL_TILE_TYPE INT_TYPE shared_localW[WORKGROUP_SIZE][2*TILE_LEN*TILE_LEN];
    //size_t lid = get_local_id(0);
    size_t lid = threadIdx.x;
    LOCAL_TILE_PTR localW = &shared_localW[lid][0];
#endif

    //size_t gid = get_global_id(0);
    size_t gid = blockIdx.x * blockDim.x + threadIdx.x;
    
    
    INT_TYPE d0 = r0 - gid*2*tileLen; 
    INT_TYPE m_top = max2(m_m,m_n);

    // printf("\n[POCL] d0=%ld r0=%ld  cv=%ld\n", d0, r0, p_final_d_r[0]);

    int doRun = 1;
    
    if (abs(d0) > r0)
        doRun = 0;
    
    if (!doRun)
        return;
        
    // we already reached the final point in previous invocations
    if (p_final_d_r[0] >= m_top)
        return;
        
    // Increase
    for (int i=0 ; i < tileLen; i++)
        for (int j=-i; ((j <= i)  && (doRun)); j++)
            processCell(P, T, m_m, m_n, m_k, m_W, p_final_d_r, d0+j, r0+i, tileLen, localW, j, i, &doRun);
    
    // Decrease
    for (int i=0 ; i < tileLen; i++)
    {
        int ii = tileLen - 1 -i;
        
        for (int j=-ii; ((j <= ii)  && (doRun)); j++)
            processCell(P, T, m_m, m_n, m_k, m_W, p_final_d_r, d0+j, r0+tileLen+i, tileLen, localW, j, tileLen+i, &doRun);
    }
}

CUDAWavefrontOriginal2Cols::CUDAWavefrontOriginal2Cols()
{
    m_buf_P = NULL;
    m_buf_T = NULL;
    m_buf_W = NULL;
    
    m_W = NULL;
}


CUDAWavefrontOriginal2Cols::~CUDAWavefrontOriginal2Cols()
{
    if (m_W != NULL)
        delete [] m_W;
}

void CUDAWavefrontOriginal2Cols::setInput(const char* P, const char* T, INT_TYPE k)
{
    // this should not be allocated, we only expect a single call
    assert(m_W == NULL);
    
    m_m = strlen(P);
    m_n = strlen(T);
    m_k = k;
    m_tileLen = gTileLen;
    
    if (gTileLen != TILE_LEN)
    {
        printf("Tile length statically compiled to %d\n", TILE_LEN);
        exit(-1);
    }

    long size = (2)*(2*k+1);

    try
    {
        m_W = new INT_TYPE[size];
    }
    catch (const std::bad_alloc& e) 
    {
        printf("FAILED to allocate memory\n");
        exit(-1);
    }

    m_P = P;
    m_T = T;

    hipMalloc(&m_buf_P,  m_m * sizeof(char));
    hipMalloc(&m_buf_T, m_n * sizeof(char));

    printf("creating buffer %.2f GB\n", size*sizeof(INT_TYPE)/(1E9));

    hipMalloc(&m_buf_W, size * sizeof(INT_TYPE));
    hipMalloc(&m_buf_final_d_r, 2 * sizeof(INT_TYPE));
    
    
    printf("input set\n");
}

void CUDAWavefrontOriginal2Cols::progress(PerformanceLap& lap, INT_TYPE r, int& lastpercent, long cellsAllocated, long cellsAlive)
{
#define DECIMALS_PERCENT    1000
    if (!verbose)
        return;
    
    double elapsed = lap.stop();
    // linear model
//    double estimated = (elapsed / r) * (m_k);
//    int percent = (r*100.0*DECIMALS_PERCENT/m_k);
    // square model
    double estimated = (elapsed / (r*r)) * (m_k*m_k);
    int percent = (r*r*100.0*DECIMALS_PERCENT/(m_k*m_k));
    
    //if (percent != lastpercent)
    {
        //printf("\rcol %ld/%ld %.2f%% cells allocated: %ld alive: %ld elapsed: %d s  estimated: %d s    ", x, m_n, ((double)percent/DECIMALS_PERCENT), cellsAllocated, cellsAlive, (int) elapsed, (int) estimated );
        printf("\rr %ld/%ld %.2f%% elapsed: %d s  estimated: %d s  ", r, m_k, ((double)percent/DECIMALS_PERCENT) , (int) elapsed, (int) estimated );
    
        fflush(stdout);
        lastpercent = percent;
    }
}

/*
static long nextMultiple(long value, long multiple)
{
   return ((value + (multiple-1)) / multiple) * multiple; 
}

static long previousMultiple(long value, long multiple)
{
   return ((value - (multiple-1)) / multiple) * multiple; 
}
*/

#define NUMBER_OF_INVOCATIONS_PER_READ gEnqueuedInvocations

INT_TYPE CUDAWavefrontOriginal2Cols::getDistance()
{
    PerformanceLap lap;
    int lastpercent = -1;
    long cellsAllocated = 0;
    long cellsAlive = 0;
        
    hipMemcpy(m_buf_P, m_P, m_m * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(m_buf_T, m_T, m_n * sizeof(char), hipMemcpyHostToDevice);
    
    INT_TYPE h = 2*m_k+1;
    
    INT_TYPE final_d = CARTESIAN_TO_POLAR_D_D(m_m, m_n);
    INT_TYPE m_top = max2(m_m,m_n);

    setCommonArgs();

    m_final_d_r[0] = 0;     // furthest reaching point
    m_final_d_r[1] = m_top; // estimated distance (now, worst case)
    
    hipMemcpy(m_buf_final_d_r, m_final_d_r, 2 * sizeof(INT_TYPE), hipMemcpyHostToDevice );
    

    if (gMeasureIterationTime)
        printf("r,time\n");
        
    for (INT_TYPE r=0; r < m_k; r+= m_tileLen)
    {
        invokeKernel(r);

        if (gMeasureIterationTime)
        {
            if (((r % 1000) == 0) & (r>0))
            {
                lap.stop();
                printf("%ld, %f\n", r, lap.lap());
            }
        }        
        else
            progress(lap, r, lastpercent, cellsAllocated, cellsAlive);
        
        if ((r % NUMBER_OF_INVOCATIONS_PER_READ) == 0)
        {
            hipMemcpy(m_final_d_r, m_buf_final_d_r, 2 * sizeof(INT_TYPE), hipMemcpyDeviceToHost);
            
            if (m_final_d_r[0] >= m_top)
                return m_final_d_r[1];
        }
    }
    
    lastpercent--;
    progress(lap, m_k, lastpercent, cellsAllocated, cellsAlive);

    return m_top;
}
    
void CUDAWavefrontOriginal2Cols::setCommonArgs()
{
}

void CUDAWavefrontOriginal2Cols::invokeKernel(INT_TYPE r)
{
    INT_TYPE k = max2(m_m,m_n);
    
    INT_TYPE threads = (r/m_tileLen)+1;
    
    INT_TYPE blocks = (threads + gWorkgroupSize - 1) / gWorkgroupSize;
    INT_TYPE threadsPerBlock = gWorkgroupSize;
     
    wfo2cols<<<blocks, threadsPerBlock>>>(m_buf_P, m_buf_T, m_m, m_n, r, k, m_buf_W, m_buf_final_d_r, m_tileLen);

}

char* CUDAWavefrontOriginal2Cols::getAlignmentPath(INT_TYPE* distance)
{
    printf("Not implemented yet\n");
    exit(-1);
}
    
const char* CUDAWavefrontOriginal2Cols::getDescription()
{
    return "Wavefront Original 2 columns [ocl_local_tiles] in CUDA";
}
