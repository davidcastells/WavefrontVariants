#include "hip/hip_runtime.h"
/**
 * 
 * Copyright (C) 2021-2022, David Castells-Rufas <david.castells@uab.cat>, 
 * Universitat Autonoma de Barcelona  
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

/* 
 * File:   CUDAWavefrontOriginal2Cols.cpp
 * Author: dcr
 * 
 * Created on April 24, 2022, 11:30 AM
 */

#include "CUDAWavefrontOriginal2Cols.h"
#include "utils.h"

#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <string.h>
#include <new>


#define CARTESIAN_TO_INDEX(y, x, w)		((y)*(w) + (x))
//#define POLAR_D_TO_INDEX(d, r, w)			CARTESIAN_TO_INDEX(POLAR_D_TO_CARTESIAN_Y((d), (r)),POLAR_D_TO_CARTESIAN_X((d), (r)),w)
#define POLAR_D_TO_CARTESIAN_Y(d,r)		((((d) >= 0)? 0: -(d)) + (r))
#define POLAR_D_TO_CARTESIAN_X(d,r)		((((d) >= 0)? (d): 0) + (r))

#define CARTESIAN_TO_POLAR_D_D(y, x)		((x)-(y))
#define CARTESIAN_TO_POLAR_D_R(y, x)		(((y)>(x))? (x) : (y))

//#define POLAR_W_TO_INDEX(d, r, w)			CARTESIAN_TO_INDEX(POLAR_W_TO_CARTESIAN_Y((d), (r)),POLAR_W_TO_CARTESIAN_X((d), (r)),w)

#define POLAR_W_TO_INDEX(d, r)		((d)+m_k + (((r)%(2*tileLen)) * (2*m_k+1)))
#define INDEX_TO_POLAR_W_D(idx, r)      ((idx) - (m_k) - ((r)%(2*tileLen))*(2*m_k+1))


#define POLAR_W_TO_CARTESIAN_Y(d,r)		((((d) >= 0)? -(d) : 0 ) + (r))
#define POLAR_W_TO_CARTESIAN_X(d,r)		((((d) >= 0)? 0 : (d)) + (r))

#define CARTESIAN_TO_POLAR_W_D(y, x)		((x)-(y))
#define CARTESIAN_TO_POLAR_W_R(y, x)		(((y)>(x))? (y) : (x))

extern int verbose;
extern int gPid;
extern int gDid;
extern int gTileLen;
extern int gMeasureIterationTime;
extern int gEnqueuedInvocations;
extern int gWorkgroupSize;

__forceinline__ __device__
long extend(const char* P, const char* T, long m, long n, long pi, long ti)
{
    long e = 0;

    while (pi < m && ti < n)
    {
            if (P[pi] != T[ti])
                    return e;
            e++;
            pi++;
            ti++;
    }

    return e;
}

__forceinline__ __device__
int polarExistsInW(long d, long r)
{
    long x = POLAR_W_TO_CARTESIAN_X(d,r);
    long y = POLAR_W_TO_CARTESIAN_Y(d,r);
	
    return ((x >= 0) && (y >= 0));
}

__forceinline__ __device__
void processCell(char* P, 
        char* T, 
        long m_m, 
        long m_n,
        long m_k, 
        long* m_W,
        long* p_final_d_r,
        long d,
        long r,
        int tileLen)
{
    long m_top = max2(m_m,m_n);

    // we already reached the final point in previous invocations
    if (p_final_d_r[0] >= m_top)
        return;
        
    long final_d = CARTESIAN_TO_POLAR_D_D(m_m, m_n);

    // early exit for useless work items
    if (!polarExistsInW(d,r))
        return;
            
    if (r == 0)
    {
        if (d == 0)
            // initial case
            m_W[POLAR_W_TO_INDEX(d, r)] = extend(P, T, m_m, m_n, 0, 0);
        else
            m_W[POLAR_W_TO_INDEX(d, r)]  = 0;
            
        // printf("W[d:%ld,r:%ld]=%ld\n", d, r, m_W[POLAR_W_TO_INDEX(d, r)]);
    }
    else
    {
        long diag_up = (polarExistsInW(d+1, r-1))? m_W[POLAR_W_TO_INDEX(d+1, r-1)]  : 0;
        long left = (polarExistsInW(d,r-1))? m_W[POLAR_W_TO_INDEX(d, r-1)]  : 0;
        long diag_down = (polarExistsInW(d-1,r-1))? m_W[POLAR_W_TO_INDEX(d-1, r-1)]  : 0;

        long compute;

        if (d == 0)
            compute = max3(diag_up, left+1, diag_down);
        else if (d > 0)
            compute = max3(diag_up, left+1, diag_down+1);
        else
            compute = max3(diag_up+1, left+1, diag_down);

        if ((d == final_d) && compute >= m_top)
        {
            m_W[POLAR_W_TO_INDEX(d, r)] = compute;
            p_final_d_r[0] = compute;   // furthest reaching point
            p_final_d_r[1] = r;         // at edit distance = r
            
            // printf("W[d:%ld,r:%ld]=%ld\n", d, r, compute);
            return;
        }

        long ex = POLAR_W_TO_CARTESIAN_X(d, compute);
        long ey = POLAR_W_TO_CARTESIAN_Y(d, compute);

        if ((ex < m_n) && (ey < m_m))
        {
            long extendv = extend(P, T, m_m, m_n, ey, ex);
            long extended = compute + extendv;

            m_W[POLAR_W_TO_INDEX(d, r)] = extended;

            if ((d == final_d) && extended >= m_top)
            {
                p_final_d_r[0] = extended;  // furthest reaching point
                p_final_d_r[1] = r;         // at edit distance = r

                //printf("W[d:%ld,r:%ld]=%ld\n", d, r, extended);
                return;
            }
        }
        else
        {
            m_W[POLAR_W_TO_INDEX(d, r)] = compute;
            // it is impossible to assign the final result here, because it would
            // have been in the previous compute check
        }
    }

}

__global__
void wfo2cols(
        char* P, 
        char* T, 
        long m_m, 
        long m_n, 
        long r0, 
        long m_k,  
        long* m_W,
        long* p_final_d_r,
        int tileLen)
{
    //size_t gid = get_global_id(0);
    size_t gid = blockIdx.x * blockDim.x + threadIdx.x;
    
    
    //long d = gid - (r-1);
    long d0 = r0 - gid*2*tileLen; 
    long m_top = max2(m_m,m_n);

    // printf("\n[POCL] d0=%ld r0=%ld  cv=%ld\n", d0, r0, p_final_d_r[0]);

    
    // we already reached the final point in previous invocations
    if (p_final_d_r[0] >= m_top)
        return;
        
    // Increase
    for (int i=0 ; i < tileLen; i++)
        for (int j=-i; j <= i; j++)
            processCell(P, T, m_m, m_n, m_k, m_W, p_final_d_r, d0+j, r0+i, tileLen);
    
    // Decrease
    for (int i=0 ; i < tileLen; i++)
    {
        int ii = tileLen - 1 -i;
        
        for (int j=-ii; j <= ii; j++)
            processCell(P, T, m_m, m_n, m_k, m_W, p_final_d_r, d0+j, r0+tileLen+i, tileLen);
    }
}

CUDAWavefrontOriginal2Cols::CUDAWavefrontOriginal2Cols()
{
    m_buf_P = NULL;
    m_buf_T = NULL;
    m_buf_W = NULL;
    
    m_W = NULL;
}


CUDAWavefrontOriginal2Cols::~CUDAWavefrontOriginal2Cols()
{
    if (m_W != NULL)
        delete [] m_W;
}

void CUDAWavefrontOriginal2Cols::setInput(const char* P, const char* T, long k)
{
    // this should not be allocated, we only expect a single call
    assert(m_W == NULL);
    
    m_m = strlen(P);
    m_n = strlen(T);
    m_k = k;
    m_tileLen = gTileLen;

    long size = (2*m_tileLen)*(2*k+1);

    try
    {
        m_W = new long[size];
    }
    catch (const std::bad_alloc& e) 
    {
        printf("FAILED to allocate memory\n");
        exit(-1);
    }

    m_P = P;
    m_T = T;

    hipMalloc(&m_buf_P,  m_m * sizeof(char));
    hipMalloc(&m_buf_T, m_n * sizeof(char));

    printf("creating buffer %.2f GB\n", size*sizeof(long)/(1E9));

    hipMalloc(&m_buf_W, size * sizeof(long));
    hipMalloc(&m_buf_final_d_r, 2 * sizeof(long));
    
    
    printf("input set\n");
}

void CUDAWavefrontOriginal2Cols::progress(PerformanceLap& lap, long r, int& lastpercent, long cellsAllocated, long cellsAlive)
{
#define DECIMALS_PERCENT    1000
    if (!verbose)
        return;
    
    double elapsed = lap.stop();
    // linear model
//    double estimated = (elapsed / r) * (m_k);
//    int percent = (r*100.0*DECIMALS_PERCENT/m_k);
    // square model
    double estimated = (elapsed / (r*r)) * (m_k*m_k);
    int percent = (r*r*100.0*DECIMALS_PERCENT/(m_k*m_k));
    
    //if (percent != lastpercent)
    {
        //printf("\rcol %ld/%ld %.2f%% cells allocated: %ld alive: %ld elapsed: %d s  estimated: %d s    ", x, m_n, ((double)percent/DECIMALS_PERCENT), cellsAllocated, cellsAlive, (int) elapsed, (int) estimated );
        printf("\rr %ld/%ld %.2f%% elapsed: %d s  estimated: %d s  ", r, m_k, ((double)percent/DECIMALS_PERCENT) , (int) elapsed, (int) estimated );
    
        fflush(stdout);
        lastpercent = percent;
    }
}

/*
static long nextMultiple(long value, long multiple)
{
   return ((value + (multiple-1)) / multiple) * multiple; 
}

static long previousMultiple(long value, long multiple)
{
   return ((value - (multiple-1)) / multiple) * multiple; 
}
*/

#define NUMBER_OF_INVOCATIONS_PER_READ gEnqueuedInvocations

long CUDAWavefrontOriginal2Cols::getDistance()
{
    PerformanceLap lap;
    int lastpercent = -1;
    long cellsAllocated = 0;
    long cellsAlive = 0;
        
    hipMemcpy(m_buf_P, m_P, m_m * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(m_buf_T, m_T, m_n * sizeof(char), hipMemcpyHostToDevice);
    
    long h = 2*m_k+1;
    
    long final_d = CARTESIAN_TO_POLAR_D_D(m_m, m_n);
    long m_top = max2(m_m,m_n);

    setCommonArgs();

    m_final_d_r[0] = 0;     // furthest reaching point
    m_final_d_r[1] = m_top; // estimated distance (now, worst case)
    
    hipMemcpy(m_buf_final_d_r, m_final_d_r, 2 * sizeof(long), hipMemcpyHostToDevice );
    

    if (gMeasureIterationTime)
        printf("r,time\n");
        
    for (long r=0; r < m_k; r+= m_tileLen)
    {
        invokeKernel(r);

        if (gMeasureIterationTime)
        {
            if (((r % 1000) == 0) & (r>0))
            {
                lap.stop();
                printf("%ld, %f\n", r, lap.lap());
            }
        }        
        else
            progress(lap, r, lastpercent, cellsAllocated, cellsAlive);
        
        if ((r % NUMBER_OF_INVOCATIONS_PER_READ) == 0)
        {
            hipMemcpy(m_final_d_r, m_buf_final_d_r, 2 * sizeof(long), hipMemcpyDeviceToHost);
            
            if (m_final_d_r[0] >= m_top)
                return m_final_d_r[1];
        }
    }
    
    lastpercent--;
    progress(lap, m_k, lastpercent, cellsAllocated, cellsAlive);

    return m_top;
}
    
void CUDAWavefrontOriginal2Cols::setCommonArgs()
{
}

void CUDAWavefrontOriginal2Cols::invokeKernel(long r)
{
    long k = max2(m_m,m_n);
    
    long threads = (r/m_tileLen)+1;
    
    long blocks = (threads + gWorkgroupSize - 1) / gWorkgroupSize;
    long threadsPerBlock = gWorkgroupSize;
     
    wfo2cols<<<blocks, threadsPerBlock>>>(m_buf_P, m_buf_T, m_m, m_n, r, k, m_buf_W, m_buf_final_d_r, m_tileLen);

}

char* CUDAWavefrontOriginal2Cols::getAlignmentPath(long* distance)
{
    printf("Not implemented yet\n");
    exit(-1);
}
    
const char* CUDAWavefrontOriginal2Cols::getDescription()
{
    return "Wavefront Original 2 columns [ocl_tiles] in OpenCL";
}
