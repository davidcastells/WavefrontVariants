#include "hip/hip_runtime.h"
/**
 * 
 * Copyright (C) 2021, David Castells-Rufas <david.castells@uab.cat>, 
 * Universitat Autonoma de Barcelona  
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

/* 
 * File:   CUDAWavefrontDynamicDiamond2Cols.cpp
 * Author: dcr
 * 
 * Created on January 14, 2022, 11:49 AM
 */

#include "CUDAWavefrontDynamicDiamond2Cols.h"
#include "utils.h"


#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <string.h>
#include <new>

#define CARTESIAN_TO_INDEX(y, x, w)		((y)*(w) + (x))
//#define POLAR_D_TO_INDEX(d, r, w)			CARTESIAN_TO_INDEX(POLAR_D_TO_CARTESIAN_Y((d), (r)),POLAR_D_TO_CARTESIAN_X((d), (r)),w)
#define POLAR_D_TO_CARTESIAN_Y(d,r)		((((d) >= 0)? 0: -(d)) + (r))
#define POLAR_D_TO_CARTESIAN_X(d,r)		((((d) >= 0)? (d): 0) + (r))

#define CARTESIAN_TO_POLAR_D_D(y, x)		((x)-(y))
#define CARTESIAN_TO_POLAR_D_R(y, x)		(((y)>(x))? (x) : (y))

//#define POLAR_W_TO_INDEX(d, r, w)			CARTESIAN_TO_INDEX(POLAR_W_TO_CARTESIAN_Y((d), (r)),POLAR_W_TO_CARTESIAN_X((d), (r)),w)

#define POLAR_W_TO_INDEX(d, r)		((d)+m_k + (((r)%(2*tileLen)) * (2*m_k+1)))

#define INDEX_TO_POLAR_W_D(idx, r)      ((idx) - (m_k) - ((r)%(2*tileLen))*(2*m_k+1))


#define POLAR_W_TO_CARTESIAN_Y(d,r)		((((d) >= 0)? -(d) : 0 ) + (r))
#define POLAR_W_TO_CARTESIAN_X(d,r)		((((d) >= 0)? 0 : (d)) + (r))

#define CARTESIAN_TO_POLAR_W_D(y, x)		((x)-(y))
#define CARTESIAN_TO_POLAR_W_R(y, x)		(((y)>(x))? (y) : (x))

/////////////////////////////////////////////////////////////////////////////////////////////////


#define POLAR_LOCAL_W_TO_INDEX(d, r, tl) ((r) >= (tl))? (2*((r)-(tl))*(tl)) - ((r)-(tl))*((r)-(tl)) + (2*(tl)-(r)-1) + (d) + (tl)*(tl) : ((r)*(r)+(r)+d)

#define max2(a,b) (((a)>(b))?(a):(b))
#define max3(a,b,c) max2(a, max2(b, c))

#define LOCAL_STORE     
#define GLOBAL_STORE    

/**
 * 
 * @param ld local diagonal coordinate
 * @param lr local radius coordinate
 * @param tileLen
 * @return 
 */
__forceinline__ __device__ 
int isInLocalBlock(int ld, int lr, int tileLen)
{
    if (lr < 0)
        return 0;
    
    // first check that there are in bounds
    if (lr >= tileLen)
    {
        // decreasing tile 
        int dec_lr = 2*tileLen - lr - 1;
        
        if (abs(ld) > dec_lr)
            return 0;
    }
    else
    {
        if (abs(ld) > lr)
            return 0;
    }
    
    // return whether the cell is in the local memory 
    int idx = POLAR_LOCAL_W_TO_INDEX(ld, lr, tileLen);
    return ((idx >= 0) && (idx <= 2*tileLen*tileLen));
}

__forceinline__ __device__
int isInLocalBlockBoundary(int ld, int lr, int tileLen)
{
    int maxd = 2*tileLen-lr-1;
    if (abs(ld) == maxd)
        return 1;
    if ((ld >= 0) && (abs(ld+1) == maxd))
        return 1;
    if ((ld <= 0) && (abs(ld-1) == maxd))
        return 1;
    
    return 0;
}

__forceinline__ __device__
long extendUnaligned(GLOBAL_STORE const char* P, 
            GLOBAL_STORE const char* T, long m, long n, long pi, long ti)
{
    long e = 0;

    while (pi < m && ti < n)
    {
	char pc = __ldg(&P[pi]);
	char tc = __ldg(&T[ti]);

            if (pc != tc)
                    return e;
            e++;
            pi++;
            ti++;
    }

    return e;
}

#define ctz(x) (__ffsll(x) - 1)

#define ALIGN_MASK 0xFFFFFFFFFFFFFFF8
 
__forceinline__ __device__
long extendAligned(GLOBAL_STORE const char* P, 
        GLOBAL_STORE const char* T, long m, long n, long pi, long ti)
{
    int pbv; // P valid bytes
    int tbv; // T valid bytes
    
    long pai;
    long tai;
    
    int pbidx;
    int tbidx;
    
    long PV;    // P value
    long TV;    // T value

    int mbv;
    unsigned long mask;
    int neq;
    
    long e = 0;
    
    //long gt = extend(P, T, m, n, pi, ti);
    
loop:
    pai = pi & ALIGN_MASK;
    tai = ti & ALIGN_MASK;
    
    PV = *(GLOBAL_STORE long*)(&P[pai]);
    TV = *(GLOBAL_STORE long*)(&T[tai]);
    
//    printf("pi: %ld ti: %ld pai: %ld tai: %ld \n", pi, ti, pai, tai);
//    printf("PV = 0x%016lX\n", PV);
//    printf("TV = 0x%016lX\n", TV);
    
    pbidx = (pi%8);
    tbidx = (ti%8);
    
    pbv = 8 - pbidx;
    tbv = 8 - tbidx;
    
    if (pbv > (m-pi)) pbv = m-pi;
    if (tbv > (n-ti)) tbv = n-pi;

    mbv = min2(pbv, tbv);    // minimum valid bytes
    
    //assert(mbv);
    if (mbv > 0)
    {
//        mask = (-1L);
//        mask <<= (mbv*8);
//        if (mbv == 8) mask = 0;

//        printf("%d -> %016lX\n", mbv, mask);
    //    
        switch (mbv)
        {
            case 1: mask = 0xFFFFFFFFFFFFFF00; break;
            case 2: mask = 0xFFFFFFFFFFFF0000; break;
            case 3: mask = 0xFFFFFFFFFF000000; break;
            case 4: mask = 0xFFFFFFFF00000000; break;
            case 5: mask = 0xFFFFFF0000000000; break;
            case 6: mask = 0xFFFF000000000000; break;
            case 7: mask = 0xFF00000000000000; break;
            case 8: mask = 0x0000000000000000; break;
        }

//        printf("pbv = %d\n", pbv);
//        printf("tbv = %d\n", tbv);

        PV = PV >> (pbidx*8);
        TV = TV >> (tbidx*8);

//        printf("PV = 0x%016lX\n", PV);
//        printf("TV = 0x%016lX\n", TV);
//        printf("MK = 0x%016lX\n", mask);

        neq = ctz((PV ^ TV) | mask) / 8;

//        printf("neq = %d\n", neq);

        e += neq;

        if (neq == mbv)
        {
            ti += neq;
            pi += neq;
            goto loop;
        }
    }
    
//    if (gStats) collectExtendStats(e);

//    printf("e = %ld\n", e);
    
//    if (e != gt)
//    {
//        printf("ERROR at pi: %ld ti: %ld\n", pi, ti);
//        exit(0);
//    }
    return e;
}



__forceinline__ __device__
long cuda_extend(GLOBAL_STORE const char* P,
        GLOBAL_STORE const char* T, long m, long n, long pi, long ti)
{    
#ifdef EXTEND_ALIGNED
        return extendAligned(P, T, m, n, pi, ti);
#else
        return extendUnaligned(P, T, m, n, pi, ti);
#endif
}

__forceinline__ __device__
int cuda_polarExistsInW(long d, long r)
{
    long x = POLAR_W_TO_CARTESIAN_X(d,r);
    long y = POLAR_W_TO_CARTESIAN_Y(d,r);
	
    return ((x >= 0) && (y >= 0));
}


#define WRITE_W(d,r, v) writeToW(m_W, localW, (d), (r), (v), m_k, tileLen, ld, lr)
#define READ_W(d,r, ld, lr)     (readFromW(m_W, localW, (d), (r), m_k, tileLen, ld, lr))


__forceinline__ __device__
void writeToW(GLOBAL_STORE long* m_W,
        LOCAL_STORE long* localW, long d, long r, long v, long m_k, int tileLen, int ld, int lr)
{    
    int lidx = POLAR_LOCAL_W_TO_INDEX(ld, lr, tileLen);

    localW[lidx] = v;

    int inBoundary = isInLocalBlockBoundary(ld, lr, tileLen);
    
#ifdef DEBUG
    printf("WR(%ld, %ld, %d) -> local WR(%d, %d, %d) -> WR idx(%d) = %ld (in boundary: %d)\n", 
            d, r, tileLen, 
            ld, lr, tileLen, lidx, v, inBoundary);
#endif
    
    if (inBoundary)
    {
        m_W[POLAR_W_TO_INDEX(d, r)] = v;
    }
}

__forceinline__ __device__
long readFromW(GLOBAL_STORE long* m_W,
        LOCAL_STORE long* localW, long d, long r, long m_k, int tileLen, int ld, int lr)
{
    int isInLocal = isInLocalBlock(ld, lr, tileLen);
    
    if (isInLocal)
    {
        int lidx = POLAR_LOCAL_W_TO_INDEX(ld, lr, tileLen);
        long lv = localW[lidx];

#ifdef DEBUG
    
        printf("RD(%ld, %ld, %d) -> local RD(%d, %d, %d) -> RD idx(%d) = %ld\n", 
            d, r, tileLen, 
            ld, lr, tileLen, lidx, lv);
#endif
    
        return lv;
    }
    else
    {
        long gv = m_W[POLAR_W_TO_INDEX(d, r)];
        
#ifdef DEBUG
        printf("RD(%ld, %ld, %d) ->  = %ld\n", 
            d, r, tileLen, gv);
#endif    
        return gv;
    }
}

__forceinline__ __device__
void processCell(GLOBAL_STORE char* P, 
        GLOBAL_STORE char* T, 
        long m_m, 
        long m_n,
        long m_k, 
        GLOBAL_STORE long* m_W,
        GLOBAL_STORE long* p_final_d_r,
        long d,
        long r,
        int tileLen,
        LOCAL_STORE long* localW,
        int ld,
        int lr,
        int* doRun)
{
    long final_d = CARTESIAN_TO_POLAR_D_D(m_m, m_n);
    long m_top = max2(m_m,m_n);

    // early exit for useless work items
    if (!cuda_polarExistsInW(d,r))
        return;
            
    if (r == 0)
    {
        if (d == 0)
            // initial case
            WRITE_W(d, r, cuda_extend(P, T, m_m, m_n, 0, 0));
        else
            WRITE_W(d, r, 0);
    }
    else
    {
        long diag_up = (cuda_polarExistsInW(d+1, r-1))? READ_W(d+1, r-1, ld+1, lr-1) : 0;
        long left = (cuda_polarExistsInW(d,r-1))? READ_W(d, r-1, ld, lr-1) : 0;
        long diag_down = (cuda_polarExistsInW(d-1,r-1))? READ_W(d-1, r-1, ld-1, lr-1) : 0;

        long compute;

        if (d == 0)
            compute = max3(diag_up, left+1, diag_down);
        else if (d > 0)
            compute = max3(diag_up, left+1, diag_down+1);
        else
            compute = max3(diag_up+1, left+1, diag_down);

        if (d == final_d) 
        {
//            printf("COMPLETE\n");
            p_final_d_r[0] = compute;   // furthest reaching point
            p_final_d_r[1] = r;         // at edit distance = r
            
            if (compute >= m_top)
            {
                WRITE_W(d, r, compute); 
                *doRun = 0;
                return;
            }
        }

        long ex = POLAR_W_TO_CARTESIAN_X(d, compute);
        long ey = POLAR_W_TO_CARTESIAN_Y(d, compute);

        if ((ex < m_n) && (ey < m_m))
        {
            long extendv = cuda_extend(P, T, m_m, m_n, ey, ex);
            long extended = compute + extendv;

            WRITE_W(d, r, extended);

            if (d == final_d) 
            {
//                printf("COMPLETE\n");
                p_final_d_r[0] = extended;  // furthest reaching point
                p_final_d_r[1] = r;         // at edit distance = r
                
                if (extended >= m_top)
                {   
                    *doRun = 0;
                    return;
                }
            }
        }
        else
        {
            WRITE_W(d, r, compute);
            // it is impossible to assign the final result here, because it would
            // have been in the previous compute check
        }
    }

}

#define TILE_LEN_MAX 3

/**
 * This is the initial kernel version.
 * The host will create as many work items as the height of the column W
 * most of them will die soon with nothing useful to do
 * 
 * @param P the pattern
 * @param T the text
 * @param m_m the length of the pattern
 * @param m_n the length of the text
 * @param r0 initial radius of the W column to compute
 * @param m_k max number of errors we are going to cover (width of the W pyramid)
 * @param m_W memory for the 2 columns of the W pyramid
 * @param pointer to 2 values (furthest reaching radius, edit distance of the previous value)
 * @param tileLen the length of the tile. The tile will contain 2 * (n)^2, where n is the 
 *                number of columns, n > 1.
 */
__global__ 
void wfdd2cols(
        GLOBAL_STORE char* __restrict__ P, 
        GLOBAL_STORE char* __restrict__ T, 
        long m_m, 
        long m_n, 
        long r0, 
        long m_k,  
        GLOBAL_STORE long* __restrict__ m_W,
        GLOBAL_STORE long* __restrict__ p_final_d_r,
        int tileLen,
        long dstart)
{
    __shared__ long localW[2*TILE_LEN_MAX*TILE_LEN_MAX];

    size_t gid = blockIdx.x; // get_global_id(0);

    //long d = gid - (r-1);
    long d0 = dstart - gid*2*tileLen; 
    long m_top = max2(m_m,m_n);
    //long final_d = CARTESIAN_TO_POLAR_D_D(m_m, m_n);
    int doRun = 1;
    
    if (abs(d0) > r0)
        doRun = 0;

#ifdef DEBUG
    printf("\ngid: %ld - final_d: %ld run: %d r: %ld dstart: %ld d0: %ld\n", gid, final_d, doRun, r0, dstart, d0);
#endif
    // printf("\n[POCL] d0=%ld r0=%ld  cv=%ld\n", d0, r0, p_final_d_r[0]);
    
    if (!doRun)
        return;
    
    // we already reached the final point in previous invocations
    if (p_final_d_r[0] >= m_top)
        return;
        
    // Increase
    for (int i=0 ; ((i < tileLen) && (doRun)); i++)
        for (int j=-i; ((j <= i) && (doRun)); j++)
            processCell(P, T, m_m, m_n, m_k, m_W, p_final_d_r, d0+j, r0+i, tileLen, localW, j, i, &doRun);
    
    // Decrease
    for (int i=0 ; ((i < tileLen) && (doRun)); i++)
    {
        int ii = tileLen - 1 -i;
        
        for (int j=-ii; ((j <= ii) && (doRun)); j++)
            processCell(P, T, m_m, m_n, m_k, m_W, p_final_d_r, d0+j, r0+tileLen+i, tileLen, localW, j, tileLen+i, &doRun);
    }
}

extern int verbose;
extern int gPid;
extern int gDid;
extern int gWorkgroupSize;
extern int gExtendAligned;
extern int gPrintPeriod;


CUDAWavefrontDynamicDiamond2Cols::CUDAWavefrontDynamicDiamond2Cols()
{
    m_W = NULL;
}


CUDAWavefrontDynamicDiamond2Cols::~CUDAWavefrontDynamicDiamond2Cols()
{    
    if (m_W != NULL)
        delete [] m_W;

}





void CUDAWavefrontDynamicDiamond2Cols::setInput(const char* P, const char* T, long k)
{
    // this should not be allocated, we only expect a single call
    assert(m_W == NULL);
    
    m_m = strlen(P);
    m_n = strlen(T);
    m_k = k;
    m_tileLen = 3;

    long size = (2*m_tileLen)*(2*k+1);

    try
    {
        m_W = new long[size];
    }
    catch (const std::bad_alloc& e) 
    {
        printf("FAILED to allocate memory\n");
        exit(-1);
    }

//    assert(m_W);
    m_P = P;
    m_T = T;

    hipMalloc(&m_buf_P,  m_m * sizeof(char));
    hipMalloc(&m_buf_T, m_n * sizeof(char));

    printf("creating buffer %.2f GB\n", size*sizeof(long)/(1E9));

    hipMalloc(&m_buf_W, size * sizeof(long));
    hipMalloc(&m_buf_final_d_r, 2 * sizeof(long));
    
//    auto ocl = OCLUtils::getInstance();
//        
//    std::string options = "-D TILE_LEN=" + std::to_string(m_tileLen) + " ";
//
//    std::string plName = ocl->getSelectedPlatformName();
//    if (OCLUtils::contains(plName, "Portable Computing Language") && (verbose > 1))
//        options += " -D DEBUG ";
//    
//    if (gExtendAligned)
//        options += " -D EXTEND_ALIGNED ";
//    
//    ocl->createProgramFromSource("WFDD2ColsGPU.cl", options);
//    m_kernel = ocl->createKernel("wfdd2cols");
    
    printf("input set\n");
}

void CUDAWavefrontDynamicDiamond2Cols::progress(PerformanceLap& lap, long r, int& lastpercent, long cellsAllocated, long cellsAlive, long numds)
{
    static double lastPrintLap = -1;
    
    double printPeriod = (gPrintPeriod > 0)? gPrintPeriod : 0.5;
    
#define DECIMALS_PERCENT    1000
    if (!verbose)
        return;
    
    double elapsed = lap.stop();
    // linear model
//    double estimated = (elapsed / r) * (m_k);
//    int percent = (r*100.0*DECIMALS_PERCENT/m_k);
    // square model
    double estimated = (elapsed / (r*r)) * (m_k*m_k);
    int percent = (r*r*100.0*DECIMALS_PERCENT/(m_k*m_k));
    
    if (elapsed > (lastPrintLap + printPeriod))
    {
        printf((gPrintPeriod > 0)?"\n":"\r");
        //printf("\rcol %ld/%ld %.2f%% cells allocated: %ld alive: %ld elapsed: %d s  estimated: %d s    ", x, m_n, ((double)percent/DECIMALS_PERCENT), cellsAllocated, cellsAlive, (int) elapsed, (int) estimated );
        printf("r %ld/%ld %.2f%% (ds: %ld) elapsed: %d s  estimated: %d s  ", r, m_k, ((double)percent/DECIMALS_PERCENT), numds , (int) elapsed, (int) estimated );
    
        fflush(stdout);
        lastpercent = percent;
        lastPrintLap = elapsed;
    }
}

/**
 * Returns the value (y) which is the next multiple of (m) higher than (x)
 * 
 * y = m*k, so that y > x
 * 
 * @param value
 * @param multiple
 * @return 
 */
static long nextMultiple(long value, long multiple)
{
   return ((value + (multiple-1)) / multiple) * multiple; 
}

static long previousMultiple(long value, long multiple)
{
   return ((value - (multiple-1)) / multiple) * multiple; 
}

#define NUMBER_OF_INVOCATIONS_PER_READ 100

long CUDAWavefrontDynamicDiamond2Cols::getDistance()
{
    PerformanceLap lap;
    int lastpercent = -1;
    long cellsAllocated = 0;
    long cellsAlive = 0;
      
    hipMemcpy(m_buf_P, m_P, m_m * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(m_buf_T, m_T, m_n * sizeof(char), hipMemcpyHostToDevice);
    
    // this is the initial height of the pyramid
    long h = 2*m_k+1;
    
    long final_d = CARTESIAN_TO_POLAR_D_D(m_m, m_n);
    long m_top = max2(m_m,m_n); // this is the maximum possible value of the W pyramid cells

    setCommonArgs();

    m_final_d_r[0] = 0;     // furthest reaching point
    m_final_d_r[1] = m_top; // estimated distance (now, worst case)
    hipMemcpy(m_buf_final_d_r, m_final_d_r, 2 * sizeof(long), hipMemcpyHostToDevice );
    

    for (long r=0; r < m_k; r+= m_tileLen)
    {
        long dup_left = r;      // this is for sure a multiple of tile len
        long ddown_left = -r;   // this is for sure a multiple of tile len
        
        long effk = nextMultiple(m_k, m_tileLen*2) + m_tileLen;
        long eff_fd_up = nextMultiple(final_d, m_tileLen);
        long eff_fd_down = previousMultiple(final_d, m_tileLen);
        
        long dup_right = eff_fd_up + (effk-r);
        long ddown_right = eff_fd_down - (effk-r);
        
        // find the next multiple of tilelen
        if ((dup_right % m_tileLen) != 0) dup_right += m_tileLen - (dup_right % m_tileLen);
        if ((ddown_right % m_tileLen) != 0) ddown_right -= m_tileLen - (ddown_right % m_tileLen);
        
        long dup = min2(dup_left, dup_right);
        long ddown = max2(ddown_left, ddown_right);
        
        long dstart = dup;
        long numds = (dup - ddown)+1;             // number of ds 
        long numwi = (((numds-1)/2)/m_tileLen)+1;   // number of workitems
        
        invokeKernel(r, dstart, numwi);

        progress(lap, r, lastpercent, cellsAllocated, cellsAlive, numds);
        
        if ((r % NUMBER_OF_INVOCATIONS_PER_READ) == 0)
        {
            hipMemcpy(m_final_d_r, m_buf_final_d_r, 2 * sizeof(long), hipMemcpyDeviceToHost);
            
            if (verbose > 1)
            {
                printf("Check to continue\nTop: %ld\n", m_top);
                printf("Fur. reaching point: %ld\n", m_final_d_r[0]);
                printf("Edit distance: %ld\n", m_final_d_r[1]);
            }
            
            if (m_final_d_r[0] >= m_top)
                return m_final_d_r[1];  // return the distance
            else
            {
                // update the maximum error to consider, if it is lower than current one
                //printf("w value: %ld - ed: %ld\n", m_final_d_r[0], m_final_d_r[1]);
                long newk = m_top - (m_final_d_r[0] - m_final_d_r[1]);
                if (newk < m_k)
                    m_k = newk;
            }
        }
    }
    
    lastpercent--;
    progress(lap, m_k, lastpercent, cellsAllocated, cellsAlive, 0);

    hipMemcpy(m_final_d_r, m_buf_final_d_r, 2 * sizeof(long), hipMemcpyDeviceToHost);
            
    if (verbose > 1)
    {
        printf("Check to continue\nTop: %ld\n", m_top);
        printf("Fur. reaching point: %ld\n", m_final_d_r[0]);
        printf("Edit distance: %ld\n", m_final_d_r[1]);
    }

    if (m_final_d_r[0] >= m_top)
        return m_final_d_r[1];

    return m_top;
}
    
void CUDAWavefrontDynamicDiamond2Cols::setCommonArgs()
{
}

void CUDAWavefrontDynamicDiamond2Cols::invokeKernel(long r, long dstart, long numds)
{
    long k = max2(m_m,m_n);

    wfdd2cols<<<numds, gWorkgroupSize>>>(m_buf_P, m_buf_T, m_m, m_n, r, k, m_buf_W, m_buf_final_d_r, m_tileLen, dstart);

}

char* CUDAWavefrontDynamicDiamond2Cols::getAlignmentPath(long* distance)
{
    printf("Not implemented yet\n");
    exit(-1);
}
    
const char* CUDAWavefrontDynamicDiamond2Cols::getDescription()
{
    return "Wavefront Dynamic Diamond 2 columns [ocl_local_tiles] in CUDA";
}
